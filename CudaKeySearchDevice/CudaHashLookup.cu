#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <vector>

#include "KeySearchDevice.h"

#include "CudaHashLookup.h"

#include "CudaHashLookup.cuh"

#include "Logger.h"

#include "util.h"

#define MAX_TARGETS_CONSTANT_MEM 16

__constant__ unsigned int _TARGET_HASH[MAX_TARGETS_CONSTANT_MEM][5];
__constant__ unsigned int _NUM_TARGET_HASHES[1];
__constant__ unsigned int *_BLOOM_FILTER[1];
__constant__ unsigned int _BLOOM_FILTER_MASK[1];
__constant__ unsigned long long _BLOOM_FILTER_MASK64[1];

__constant__ unsigned int _USE_BLOOM_FILTER[1];


static unsigned int swp(unsigned int x)
{
	return (x << 24) | ((x << 8) & 0x00ff0000) | ((x >> 8) & 0x0000ff00) | (x >> 24);
}

static void undoRMD160FinalRound(const unsigned int hIn[5], unsigned int hOut[5])
{
	unsigned int iv[5] = {
		0x67452301,
		0xefcdab89,
		0x98badcfe,
		0x10325476,
		0xc3d2e1f0
	};

	for(int i = 0; i < 5; i++) {
		hOut[i] = swp(hIn[i]) - iv[(i + 1) % 5];
	}
}

/**
Copies the target hashes to constant memory
*/
hipError_t CudaHashLookup::setTargetConstantMemory(const std::vector<struct hash160> &targets)
{
	size_t count = targets.size();

	for(size_t i = 0; i < count; i++) {
		unsigned int h[5];

		undoRMD160FinalRound(targets[i].h, h);

		hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(_TARGET_HASH), h, sizeof(unsigned int) * 5, i * sizeof(unsigned int) * 5);

		if(err) {
			return err;
		}
	}

	hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(_NUM_TARGET_HASHES), &count, sizeof(unsigned int));
	if(err) {
		return err;
	}

	unsigned int useBloomFilter = 0;

	err = hipMemcpyToSymbol(HIP_SYMBOL(_USE_BLOOM_FILTER), &useBloomFilter, sizeof(bool));
	if(err) {
		return err;
	}

	return hipSuccess;
}

/**
Returns the optimal bloom filter size in bits given the probability of false-positives and the
number of hash functions
*/
unsigned int CudaHashLookup::getOptimalBloomFilterBits(double p, size_t n)
{
	double m = 3.6 * ceil((n * log(p)) / log(1 / pow(2, log(2))));

	return (unsigned int)ceil(log(m) / log(2));
}

void CudaHashLookup::initializeBloomFilter(const std::vector<struct hash160> &targets, unsigned int *filter, unsigned int mask)
{
	// Use the low 16 bits of each word in the hash as the index into the bloom filter
	for(unsigned int i = 0; i < targets.size(); i++) {

		unsigned int h[5];

		undoRMD160FinalRound(targets[i].h, h);

		for(int j = 0; j < 5; j++) {
			unsigned int idx = h[j] & mask;

			filter[idx / 32] |= (0x01 << (idx % 32));
		}

	}
}

void CudaHashLookup::initializeBloomFilter64(const std::vector<struct hash160> &targets, unsigned int *filter, unsigned long long mask)
{
	for(unsigned int k = 0; k < targets.size(); k++) {

		unsigned int hash[5];

		unsigned long long idx[5];

		undoRMD160FinalRound(targets[k].h, hash);

		idx[0] = ((unsigned long long)hash[0] << 32 | hash[1]) & mask;
		idx[1] = ((unsigned long long)hash[2] << 32 | hash[3]) & mask;
		idx[2] = ((unsigned long long)(hash[0]^hash[1]) << 32 | (hash[1]^hash[2])) & mask;
		idx[3] = ((unsigned long long)(hash[2]^hash[3]) << 32 | (hash[3] ^ hash[4])) & mask;
		idx[4] = ((unsigned long long)(hash[0]^hash[3]) << 32 | (hash[1]^hash[3])) & mask;

		for(int i = 0; i < 5; i++) {

			filter[idx[i] / 32] |= (0x01 << (idx[i] % 32));
		}
	}
}

/**
Populates the bloom filter with the target hashes
*/
hipError_t CudaHashLookup::setTargetBloomFilter(const std::vector<struct hash160> &targets)
{
	unsigned int bloomFilterBits = getOptimalBloomFilterBits(1.0e-9, targets.size());

	unsigned long long bloomFilterSizeWords = (unsigned long long)1 << (bloomFilterBits - 5);
	unsigned long long bloomFilterBytes = (unsigned long long)1 << (bloomFilterBits - 3);
	unsigned long long bloomFilterMask = (((unsigned long long)1 << bloomFilterBits) - 1);

	Logger::log(LogLevel::Info, "Allocating bloom filter (" + util::format("%.1f", (double)bloomFilterBytes/(double)(1024*1024)) + "MB)");

	unsigned int *filter = NULL;
	
	try {
		filter = new unsigned int[bloomFilterSizeWords];
	} catch(std::bad_alloc) {
		Logger::log(LogLevel::Error, "Out of system memory");

		return hipErrorOutOfMemory;
	}

	hipError_t err = hipMalloc(&_bloomFilterPtr, bloomFilterBytes);

	if(err) {
		Logger::log(LogLevel::Error, "Device error: " + std::string(hipGetErrorString(err)));
		delete[] filter;
		return err;
	}

	memset(filter, 0, sizeof(unsigned int) * bloomFilterSizeWords);
	if(bloomFilterBits > 32) {
		initializeBloomFilter64(targets, filter, bloomFilterMask);
	} else {
		initializeBloomFilter(targets, filter, (unsigned int)bloomFilterMask);
	}

	// Copy to device
	err = hipMemcpy(_bloomFilterPtr, filter, sizeof(unsigned int) * bloomFilterSizeWords, hipMemcpyHostToDevice);
	if(err) {
		hipFree(_bloomFilterPtr);
		_bloomFilterPtr = NULL;
		delete[] filter;
		return err;
	}

	// Copy device memory pointer to constant memory
	err = hipMemcpyToSymbol(HIP_SYMBOL(_BLOOM_FILTER), &_bloomFilterPtr, sizeof(unsigned int *));
	if(err) {
		hipFree(_bloomFilterPtr);
		_bloomFilterPtr = NULL;
		delete[] filter;
		return err;
	}

	// Copy device memory pointer to constant memory
	if(bloomFilterBits <= 32) {
		err = hipMemcpyToSymbol(HIP_SYMBOL(_BLOOM_FILTER_MASK), &bloomFilterMask, sizeof(unsigned int *));
		if(err) {
			hipFree(_bloomFilterPtr);
			_bloomFilterPtr = NULL;
			delete[] filter;
			return err;
		}
	} else {
		err = hipMemcpyToSymbol(HIP_SYMBOL(_BLOOM_FILTER_MASK64), &bloomFilterMask, sizeof(unsigned long long *));
		if(err) {
			hipFree(_bloomFilterPtr);
			_bloomFilterPtr = NULL;
			delete[] filter;
			return err;
		}
	}

	unsigned int useBloomFilter = bloomFilterBits <= 32 ? 1 : 2;

	err = hipMemcpyToSymbol(HIP_SYMBOL(_USE_BLOOM_FILTER), &useBloomFilter, sizeof(unsigned int));

	delete[] filter;

	return err;
}

/**
*Copies the target hashes to either constant memory, or the bloom filter depending
on how many targets there are
*/
hipError_t CudaHashLookup::setTargets(const std::vector<struct hash160> &targets)
{
	cleanup();

	if(targets.size() <= MAX_TARGETS_CONSTANT_MEM) {
		return setTargetConstantMemory(targets);
	} else {
		return setTargetBloomFilter(targets);
	}
}

void CudaHashLookup::cleanup()
{
	if(_bloomFilterPtr != NULL) {
		hipFree(_bloomFilterPtr);
		_bloomFilterPtr = NULL;
	}
}

__device__ bool checkBloomFilter(const unsigned int hash[5])
{
	bool foundMatch = true;

	unsigned int mask = _BLOOM_FILTER_MASK[0];
	unsigned int *bloomFilter = _BLOOM_FILTER[0];

	for(int i = 0; i < 5; i++) {
        unsigned int idx = hash[i] & mask;

        unsigned int f = bloomFilter[idx / 32];

		if((f & (0x01 << (idx % 32))) == 0) {
			foundMatch = false;
		}
	}

	return foundMatch;
}

__device__ bool checkBloomFilter64(const unsigned int hash[5])
{
	bool foundMatch = true;

	unsigned long long mask = _BLOOM_FILTER_MASK64[0];
	unsigned int *bloomFilter = _BLOOM_FILTER[0];
	unsigned long long idx[5];

	idx[0] = ((unsigned long long)hash[0] << 32 | hash[1]) & mask;
	idx[1] = ((unsigned long long)hash[2] << 32 | hash[3]) & mask;
	idx[2] = ((unsigned long long)(hash[0] ^ hash[1]) << 32 | (hash[1] ^ hash[2])) & mask;
	idx[3] = ((unsigned long long)(hash[2] ^ hash[3]) << 32 | (hash[3] ^ hash[4])) & mask;
	idx[4] = ((unsigned long long)(hash[0] ^ hash[3]) << 32 | (hash[1] ^ hash[3])) & mask;

	for(int i = 0; i < 5; i++) {
		unsigned int f = bloomFilter[idx[i] / 32];

		if((f & (0x01 << (idx[i] % 32))) == 0) {
			foundMatch = false;
		}
	}

	return foundMatch;
}


__device__ bool checkHash(const unsigned int hash[5])
{
	bool foundMatch = false;

	if(*_USE_BLOOM_FILTER == 1) {
		return checkBloomFilter(hash);
	} else if(*_USE_BLOOM_FILTER == 2) {
		return checkBloomFilter64(hash);
	} else {
		for(int j = 0; j < *_NUM_TARGET_HASHES; j++) {
			bool equal = true;
			for(int i = 0; i < 5; i++) {
				equal &= (hash[i] == _TARGET_HASH[j][i]);
			}

			foundMatch |= equal;
		}
	}

	return foundMatch;
}