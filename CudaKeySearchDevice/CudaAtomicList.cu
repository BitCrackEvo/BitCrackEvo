#include "hip/hip_runtime.h"
#include "CudaAtomicList.h"
#include "CudaAtomicList.cuh"

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

static __constant__ void *_LIST_BUF[1];
static __constant__ unsigned int *_LIST_SIZE[1];


__device__ void atomicListAdd(void *info, unsigned int size)
{
	unsigned int count = atomicAdd(_LIST_SIZE[0], 1);

	unsigned char *ptr = (unsigned char *)(_LIST_BUF[0]) + count * size;

	memcpy(ptr, info, size);
}

static hipError_t setListPtr(void *ptr, unsigned int *numResults)
{
	hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(_LIST_BUF), &ptr, sizeof(void *));

	if(err) {
		return err;
	}

	err = hipMemcpyToSymbol(HIP_SYMBOL(_LIST_SIZE), &numResults, sizeof(unsigned int *));

	return err;
}


hipError_t CudaAtomicList::init(unsigned int itemSize, unsigned int maxItems)
{
	_itemSize = itemSize;

	// The number of results found in the most recent kernel run
	_countHostPtr = NULL;
	hipError_t err = hipHostAlloc(&_countHostPtr, sizeof(unsigned int), hipHostMallocMapped);
	if(err) {
		goto end;
	}

	// Number of items in the list
	_countDevPtr = NULL;
	err = hipHostGetDevicePointer(&_countDevPtr, _countHostPtr, 0);
	if(err) {
		goto end;
	}
	*_countHostPtr = 0;

	// Storage for results data
	_hostPtr = NULL;
	err = hipHostAlloc(&_hostPtr, itemSize * maxItems, hipHostMallocMapped);
	if(err) {
		goto end;
	}

	// Storage for results data (device to host pointer)
	_devPtr = NULL;
	err = hipHostGetDevicePointer(&_devPtr, _hostPtr, 0);

	if(err) {
		goto end;
	}

	err = setListPtr(_devPtr, _countDevPtr);

end:
	if(err) {
		hipHostFree(_countHostPtr);

		hipFree(_countDevPtr);

		hipHostFree(_hostPtr);

		hipFree(_devPtr);
	}

	return err;
}

unsigned int CudaAtomicList::size()
{
	return *_countHostPtr;
}

void CudaAtomicList::clear()
{
	*_countHostPtr = 0;
}

unsigned int CudaAtomicList::read(void *ptr, unsigned int count)
{
	if(count >= *_countHostPtr) {
		count = *_countHostPtr;
	}

	memcpy(ptr, _hostPtr, count * _itemSize);

	return count;
}

void CudaAtomicList::cleanup()
{
	hipHostFree(_countHostPtr);

	hipFree(_countDevPtr);

	hipHostFree(_hostPtr);

	hipFree(_devPtr);
}